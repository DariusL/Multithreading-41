#include "hip/hip_runtime.h"


#include <stdio.h>


#include <omp.h>
#include <string>
#include <fstream>
#include <vector>
#include <iomanip>
#include <iostream>
#include <sstream>

using namespace std;

struct GpuStruct
{
        char pav[50];
        int kiekis;
        double kaina;
};

class Struct
{
        string pav;
        int kiekis;
        double kaina;
        GpuStruct gpuStruct;
public:
        Struct(string input = " 0 0");
        GpuStruct GetDev(){return gpuStruct;}
        string Print();
};

Struct::Struct(string input)
{
        int start, end;
        start = 0;
        end = input.find(' ');
        pav = input.substr(0, end).c_str();
        start = end + 1;
        end = input.find(' ', start);
        kiekis = stoi(input.substr(start, end - start));
        start = end + 1;
        kaina = stod(input.substr(start));
        gpuStruct.kaina = kaina;
        gpuStruct.kiekis = kiekis;
		memcpy(gpuStruct.pav, pav.c_str(), pav.length() + 1);
}

string Struct::Print()
{
        stringstream ss;
        ss << setw(15) << pav << setw(7) << kiekis << setw(20) << kaina;
        return ss.str();
}

vector<vector<Struct>> ReadStuff(string file);
vector<string> ReadLines(string file);

string Titles();
string Print(int nr, Struct &s);
void syncOut(vector<vector<Struct>>&);

void __global__ Add(GpuStruct *data, int *starts, int arrCount, GpuStruct *res);

int main()
{
        auto input = ReadStuff("LapunasD.txt");
        int count = 0;
        //suskaiciuojama kiek is viso yra duomenu
        for(auto &vec : input)
                count += vec.size();
		int width = 0;
		for(auto &vec : input)
				width = vec.size() > width ? vec.size() : width;

        cout << "\nsinchroninis isvedimas\n\n";
        syncOut(input);
        
        //procesu duomenu pradzios indeksai
        vector<int> starts;
        //lokalios GPU strukturu kopijos
        vector<GpuStruct> localStructs;
        
        int put = 0;
        for(auto &vec : input)
        {
                //proceso pradzia
                starts.push_back(put);
                for(auto &s : vec)
                {
                        localStructs.push_back(s.GetDev());
                        put++;
                }
        }
        starts.push_back(put);
        int *startsdev;
        //pradziu masyvas GPU
        hipMalloc(&startsdev, sizeof(int) * starts.size());
		hipMemcpy(startsdev, starts.data(), sizeof(int) * starts.size(), hipMemcpyHostToDevice);
        GpuStruct *arr;
        //strukturu masyvas GPU
        hipMalloc(&arr, sizeof(GpuStruct) * count);
		hipMemcpy(arr, localStructs.data(), sizeof(GpuStruct) * count, hipMemcpyHostToDevice);

        GpuStruct* gpuRes;
		hipMalloc(&gpuRes, sizeof(GpuStruct) * width);

        Add<<<1, width>>>(arr, startsdev, input.size(), gpuRes);
        //palaukiam kol gpu baigs spausdint, "pause" uzrakina konsole
        hipDeviceSynchronize();

		GpuStruct *res = (GpuStruct*)malloc(sizeof(GpuStruct) * width);
		hipMemcpy(res, gpuRes, sizeof(GpuStruct) * width, hipMemcpyDeviceToHost);
		
        cout << "\n\n" << setw(3) << "Nr" << setw(30) << "Pavadiniams" << setw(7) << "Kiekis" << setw(10) << "Kaina" << "\n\n";
		for(int i = 0; i < width; i++)
		{
			cout << setw(3) << i << setw(30) << res[i].pav << setw(7) << res[i].kiekis << setw(10) << res[i].kaina << endl;
		}

        system("pause");
        //atlaisvinami pagrindiniai masyvai, teksto eilutes atlaisvinamos sunaikintant pagrindines strukturas - input
        hipFree(arr);
        hipFree(startsdev);
        return 0;
}

vector<vector<Struct>> ReadStuff(string file)
{
        auto lines = ReadLines(file);
        vector<vector<Struct>> ret;
        vector<Struct> tmp;
        for(size_t i = 0; i < lines.size(); i++)
        {
                if(lines[i] == "")
                {
                        ret.push_back(move(tmp));
                }
                else
                {
                        tmp.emplace_back(lines[i]);
                }
        }
        return ret;
}

vector<string> ReadLines(string file)
{
        vector<string> ret;
        ifstream duom(file);
        while(!duom.eof())
        {
                string line;
                getline(duom, line);
                ret.push_back(line);
        }
        return ret;
}

string Titles()
{
        stringstream ss;
        ss << setw(15) << "Pavadiniams" << setw(7) << "Kiekis" << setw(20) << "Kaina";
        return ss.str();
}

void syncOut(vector<vector<Struct>> &data)
{
        cout << setw(3) << "Nr" << Titles() << endl << endl;
        for(size_t i = 0; i < data.size(); i++)
        {
                auto &vec = data[i];
                cout << "Masyvas" << i << endl;
                for(size_t j = 0; j < vec.size(); j++)
                {
                        cout << Print(j, vec[j]) << endl;
                }
        }
}

string Print(int nr, Struct &s)
{
        stringstream ss;
        ss << setw(3) << nr << s.Print();
        return ss.str();
}

void __global__ Add(GpuStruct *data, int *starts, int arrCount, GpuStruct *res)
{
	int id = threadIdx.x;
	int length = 0;

	GpuStruct *myRes = res + id;
	myRes->pav[length-1] = 0;
	myRes->kaina = 0.0;
	myRes->kiekis = 0;
	int ind = 0;
	for(int i = 0; i < arrCount; i++)
	{
		if(starts[i] + id < starts[i+1])
		{
			GpuStruct *src = data + starts[i] + id;
			myRes->kaina += src->kaina;
			myRes->kiekis += src->kiekis;
			for(int j = 0; src->pav[j] != 0; j++, ind++)
			{
				myRes->pav[ind] = src->pav[j];
			}
		}
	}
	myRes->pav[ind] = 0;
}