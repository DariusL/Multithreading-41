#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <hip/hip_runtime.h>

#include <omp.h>
#include <string>
#include <fstream>
#include <vector>
#include <iomanip>
#include <iostream>
#include <sstream>

using namespace std;

struct GpuStruct
{
        char *pav;
        int kiekis;
        double kaina;
		int strlen;
		GpuStruct();
};

class Struct
{
        string pav;
        int kiekis;
        double kaina;
        GpuStruct gpuStruct;
public:
        Struct(string input = " 0 0");
		Struct();
        ~Struct(){hipFree(gpuStruct.pav);}
        GpuStruct GetDev(){return gpuStruct;}
        string Print();
};

Struct::Struct(string input)
{
        int start, end;
        start = 0;
        end = input.find(' ');
        pav = input.substr(0, end).c_str();
        start = end + 1;
        end = input.find(' ', start);
        kiekis = stoi(input.substr(start, end - start));
        start = end + 1;
        kaina = stod(input.substr(start));
        gpuStruct.kaina = kaina;
        gpuStruct.kiekis = kiekis;
		gpuStruct.strlen = pav.length();
        hipMalloc(&gpuStruct.pav, pav.size() + 1);
        hipMemcpy(gpuStruct.pav, pav.c_str(), pav.size() + 1, hipMemcpyHostToDevice);
}

string Struct::Print()
{
        stringstream ss;
        ss << setw(15) << pav << setw(7) << kiekis << setw(20) << kaina;
        return ss.str();
}

vector<vector<Struct>> ReadStuff(string file);
vector<string> ReadLines(string file);

string Titles();
string Print(int nr, Struct &s);
void syncOut(vector<vector<Struct>>&);

void __global__ Add(GpuStruct *data, int *starts, int arrCount, GpuStruct *res);

int main()
{
        auto input = ReadStuff("LapunasD.txt");
        int count = 0;
        //suskaiciuojama kiek is viso yra duomenu
        for(auto &vec : input)
                count += vec.size();
		int width = 0;
		for(auto &vec : input)
				width = vec.size() > width ? vec.size() : width;

        cout << "\nsinchroninis isvedimas\n\n";
        syncOut(input);
        cout << "\nasinchroninis isvedimas\n\n";
        cout << setw(10) << "Procesas" << setw(3) << "Nr" << Titles() << "\n\n";
        
        //procesu duomenu pradzios indeksai
        vector<int> starts;
        //lokalios GPU strukturu kopijos
        vector<GpuStruct> localStructs;
        
        int put = 0;
        for(auto &vec : input)
        {
                //proceso pradzia
                starts.push_back(put);
                for(auto &s : vec)
                {
                        localStructs.push_back(s.GetDev());
                        put++;
                }
        }
        starts.push_back(put);
        int *startsdev;
        //pradziu masyvas GPU
        hipMalloc(&startsdev, sizeof(int) * starts.size());
        hipMemcpy(startsdev, &starts[0], sizeof(int) * starts.size(), hipMemcpyHostToDevice);
        GpuStruct *arr;
        //strukturu masyvas GPU
        hipMalloc(&arr, sizeof(GpuStruct) * count);
        hipMemcpy(arr, &localStructs[0], sizeof(GpuStruct) * count, hipMemcpyHostToDevice);
        //GPU funkcija
        GpuStruct* gpuRes;

		hipMalloc(&gpuRes, sizeof(GpuStruct) * width);

        Add<<<1, width>>>(arr, startsdev, input.size(), gpuRes);
        //palaukiam kol gpu baigs spausdint, "pause" uzrakina konsole
        hipDeviceSynchronize();
        system("pause");
        //atlaisvinami pagrindiniai masyvai, teksto eilutes atlaisvinamos sunaikintant pagrindines strukturas - input
        hipFree(arr);
        hipFree(startsdev);
        return 0;

}

vector<vector<Struct>> ReadStuff(string file)
{
        auto lines = ReadLines(file);
        vector<vector<Struct>> ret;
        vector<Struct> tmp;
        for(size_t i = 0; i < lines.size(); i++)
        {
                if(lines[i] == "")
                {
                        ret.push_back(move(tmp));
                }
                else
                {
                        tmp.emplace_back(lines[i]);
                }
        }
        return ret;
}

vector<string> ReadLines(string file)
{
        vector<string> ret;
        ifstream duom(file);
        while(!duom.eof())
        {
                string line;
                getline(duom, line);
                ret.push_back(line);
        }
        return ret;
}

string Titles()
{
        stringstream ss;
        ss << setw(15) << "Pavadiniams" << setw(7) << "Kiekis" << setw(20) << "Kaina";
        return ss.str();
}

void syncOut(vector<vector<Struct>> &data)
{
        cout << setw(3) << "Nr" << Titles() << endl << endl;
        for(size_t i = 0; i < data.size(); i++)
        {
                auto &vec = data[i];
                cout << "Masyvas" << i << endl;
                for(size_t j = 0; j < vec.size(); j++)
                {
                        cout << Print(j, vec[j]) << endl;
                }
        }
}

string Print(int nr, Struct &s)
{
        stringstream ss;
        ss << setw(3) << nr << s.Print();
        return ss.str();
}

void __global__ Add(GpuStruct *data, int *starts, int arrCount, GpuStruct *res)
{
	int id = threadIdx.x;
	int length = 0;
	for(int i = 0; i < arrCount; i++)
	{
		if(starts[i] + id < starts[i+1])
			length += data[starts[i]].strlen + id;
	}
	res[id].strlen = length;
	cudaMallock(&res[id].pav, length + 1);
	res[id].pav[length] = 0;
	int ind = 0;
	for(int i = 0; i < arrCount; i++)
	{
		
	}
}