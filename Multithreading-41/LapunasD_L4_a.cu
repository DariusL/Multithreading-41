#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <hip/hip_runtime.h>

#include <omp.h>
#include <string>
#include <fstream>
#include <vector>
#include <iomanip>
#include <iostream>
#include <sstream>

using namespace std;

struct GpuStruct
{
        char *pav;
        int kiekis;
        double kaina;
		GpuStruct();
};

class Struct
{
        string pav;
        int kiekis;
        double kaina;
        GpuStruct gpuStruct;
public:
        Struct(string input = " 0 0");
		Struct();
        ~Struct(){hipFree(gpuStruct.pav);}
        GpuStruct GetDev(){return gpuStruct;}
        string Print();
};

Struct::Struct(string input)
{
        int start, end;
        start = 0;
        end = input.find(' ');
        pav = input.substr(0, end).c_str();
        start = end + 1;
        end = input.find(' ', start);
        kiekis = stoi(input.substr(start, end - start));
        start = end + 1;
        kaina = stod(input.substr(start));
        gpuStruct.kaina = kaina;
        gpuStruct.kiekis = kiekis;
        hipMalloc(&gpuStruct.pav, pav.size() + 1);
        hipMemcpy(gpuStruct.pav, pav.c_str(), pav.size() + 1, hipMemcpyHostToDevice);
}

string Struct::Print()
{
        stringstream ss;
        ss << setw(15) << pav << setw(7) << kiekis << setw(20) << kaina;
        return ss.str();
}

vector<vector<Struct>> ReadStuff(string file);
vector<string> ReadLines(string file);

string Titles();
string Print(int nr, Struct &s);
void syncOut(vector<vector<Struct>>&);

void __global__ Add(GpuStruct **data, GpuStruct *ret);

int main()
{
        auto input = ReadStuff("LapunasD.txt");
        int count = 0;
		for(auto &vec : input)
			count = vec.size() > count ? vec.size() : count;
        cout << "\nsinchroninis isvedimas\n\n";
        syncOut(input);
        cout << "\nasinchroninis isvedimas\n\n";
        cout << setw(10) << "Procesas" << setw(3) << "Nr" << Titles() << "\n\n";
        
		vector<GpuStruct*> gpuStructs;

		vector<Struct> localRes;
		GpuStruct* gpuRes;

		GpuStruct** data;

		hipMalloc(&gpuRes, sizeof(GpuStruct) * count);
		for(int i = 0; i < count; i++)
		{
			localRes.emplace_back();
			hipMemcpy(gpuRes + i * sizeof(GpuStruct), &localRes[i], sizeof(GpuStruct), hipMemcpyHostToDevice);
		}

		for(int i = 0; i < input.size(); i++)
		{
			GpuStruct *tmp;
			auto &vec = input[i];
			hipMalloc(&tmp, sizeof(GpuStruct) * vec.size());

			for(int j = 0; j < vec.size(); j++)
			{
				hipMemcpy(&tmp + j * sizeof(GpuStruct), &vec[i].GetDev(), sizeof(GpuStruct), hipMemcpyHostToDevice);
			}
			gpuStructs.push_back(tmp);
		}

		hipMalloc(&data, sizeof(GpuStruct*) * input.size());
		for(int i = 0; i < input.size(); i++)
		{
			hipMemcpy(data + sizeof(GpuStruct*) * i, gpuStructs[i], sizeof(GpuStruct*), hipMemcpyHostToDevice);
		}

		Add<<<1, input.size()>>>(data, res);

        return 0;
}

vector<vector<Struct>> ReadStuff(string file)
{
        auto lines = ReadLines(file);
        vector<vector<Struct>> ret;
        vector<Struct> tmp;
        for(size_t i = 0; i < lines.size(); i++)
        {
                if(lines[i] == "")
                {
                        ret.push_back(move(tmp));
                }
                else
                {
                        tmp.emplace_back(lines[i]);
                }
        }
        return ret;
}

vector<string> ReadLines(string file)
{
        vector<string> ret;
        ifstream duom(file);
        while(!duom.eof())
        {
                string line;
                getline(duom, line);
                ret.push_back(line);
        }
        return ret;
}

string Titles()
{
        stringstream ss;
        ss << setw(15) << "Pavadiniams" << setw(7) << "Kiekis" << setw(20) << "Kaina";
        return ss.str();
}

void syncOut(vector<vector<Struct>> &data)
{
        cout << setw(3) << "Nr" << Titles() << endl << endl;
        for(size_t i = 0; i < data.size(); i++)
        {
                auto &vec = data[i];
                cout << "Masyvas" << i << endl;
                for(size_t j = 0; j < vec.size(); j++)
                {
                        cout << Print(j, vec[j]) << endl;
                }
        }
}

string Print(int nr, Struct &s)
{
        stringstream ss;
        ss << setw(3) << nr << s.Print();
        return ss.str();
}

void __global__ Add(GpuStruct **data, GpuStruct *ret)
{
}