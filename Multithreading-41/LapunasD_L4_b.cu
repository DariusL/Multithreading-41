#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include <stdio.h>
#include <hip/hip_runtime.h>

#include <string>
#include <fstream>
#include <vector>
#include <iomanip>
#include <iostream>
#include <sstream>

using namespace std;

//string concat funkcija vaizdo plokstei
__device__ void strcat_dev(char *dst, const char *src)
{
	int i = 0;
	for(; dst[i] != 0; i++);
	for(int j = 0; src[j] != 0; j++, i++)
		dst[i] = src[j];
	dst[i] = 0;
}

//struktura keliama i gpu su atitinkamais konstruktoriais
struct Struct
{
    char pav[50];
    int kiekis;
    double kaina;
public:
	__host__ Struct(string pav, int kiekis, double kaina);
	__device__ Struct();
    string Print();
};

__host__ Struct::Struct(string pav, int kiekis, double kaina)
	:kiekis(kiekis), kaina(kaina)
{
	this->pav[0] = 0;
	strcat(this->pav, pav.c_str());
}

__device__ Struct::Struct()
	:kiekis(0), kaina(0.0)
{
	pav[0] = 0;
}

string Struct::Print()
{
        stringstream ss;
        ss << setw(15) << pav << setw(7) << kiekis << setw(20) << kaina;
        return ss.str();
}

//operatorius sudeti 2 strukturas, paleidziamas kazkur giliai thrust
__device__ Struct operator+(const Struct &left, const Struct &right)
{
	Struct ret;
	ret.kiekis = left.kiekis + right.kiekis;
	ret.kaina = left.kaina + right.kaina;
	strcat_dev(ret.pav, left.pav);
	strcat_dev(ret.pav, right.pav);
	return ret;
}

vector<vector<Struct>> ReadStuff(string file);
vector<string> ReadLines(string file);

string Titles();
string Print(int nr, Struct &s);
void syncOut(vector<vector<Struct>>&);


int main()
{
    auto data = ReadStuff("LapunasD.txt");
	syncOut(data);
	vector<Struct> flatData;
	vector<int> keys;

	//duomenys suplojami i viena masyva, sudaromas raktu masyvas
	int width = 0;
	for(int i = 0; i < data.size(); i++)
	{
		width = data[i].size() > width ? data[i].size() : width;
		for(int j = 0; j < data[i].size(); j++)
		{
			keys.push_back(j);
			flatData.push_back(data[i][j]);
		}
	}
	
    thrust::equal_to<int> binary_pred;
    thrust::plus<Struct> binary_op;
    //rikiuojama pagal raktus
	thrust::sort_by_key(keys.data(), keys.data() + keys.size(), flatData.data());

	//daug castinimo
	thrust::host_vector<int> host_keys = keys;
	thrust::device_vector<int> device_keys = host_keys;

	thrust::host_vector<Struct> host_values = flatData;
	thrust::device_vector<Struct> device_values = host_values;

	thrust::device_vector<Struct> output_values;
	output_values.reserve(flatData.size());
	thrust::device_vector<int> output_keys;
	output_keys.reserve(flatData.size());

	//sumavimas pagal raktus
	thrust::reduce_by_key(device_keys.begin(), device_keys.end(), device_values.begin(), output_keys.begin(), output_values.begin(), binary_pred, binary_op);

	//cia kazkam naudojau
	thrust::host_vector<int> result_keys = output_keys;
	
	cout << "\n\n" << setw(3) << "Nr" << setw(30) << "Pavadiniams" << setw(7) << "Kiekis" << setw(10) << "Kaina" << "\n\n";
	for(int i = 0; i < width; i++)
	{
		//spausdinami sudeti rezultatai
		Struct res = output_values[i];
		cout << setw(3) << i << setw(30) << res.pav << setw(7) << res.kiekis << setw(10) << res.kaina << endl;
	}
	system("pause");
	return 0;
}

vector<vector<Struct>> ReadStuff(string file)
{
    auto lines = ReadLines(file);
    vector<vector<Struct>> ret;
    vector<Struct> tmp;
    for(size_t i = 0; i < lines.size(); i++)
    {
        if(lines[i] == "")
        {
            ret.push_back(move(tmp));
        }
        else
        {
			int start, end;
			start = 0;
			end = lines[i].find(' ');
			string pav = lines[i].substr(0, end);
			start = end + 1;
			end = lines[i].find(' ', start);
			int kiekis = stoi(lines[i].substr(start, end - start));
			start = end + 1;
			double kaina = stod(lines[i].substr(start));
			tmp.emplace_back(pav.c_str(), kiekis, kaina);
        }
    }
    return ret;
}

vector<string> ReadLines(string file)
{
        vector<string> ret;
        ifstream duom(file);
        while(!duom.eof())
        {
                string line;
                getline(duom, line);
                ret.push_back(line);
        }
        return ret;
}

string Titles()
{
        stringstream ss;
        ss << setw(15) << "Pavadiniams" << setw(7) << "Kiekis" << setw(20) << "Kaina";
        return ss.str();
}

void syncOut(vector<vector<Struct>> &data)
{
        cout << setw(3) << "Nr" << Titles() << endl << endl;
        for(size_t i = 0; i < data.size(); i++)
        {
                auto &vec = data[i];
                cout << "Masyvas" << i << endl;
                for(size_t j = 0; j < vec.size(); j++)
                {
                        cout << Print(j, vec[j]) << endl;
                }
        }
}

string Print(int nr, Struct &s)
{
        stringstream ss;
        ss << setw(3) << nr << s.Print();
        return ss.str();
}