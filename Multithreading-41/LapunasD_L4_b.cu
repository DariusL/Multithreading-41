//Darius Lapunas, IFF-1, 15 kompiuteris
/*
	Pakeista:
	77
	122-126
	140-146
	152-155
	243-247
*/

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include <stdio.h>
#include <hip/hip_runtime.h>

#include <omp.h>
#include <string>
#include <fstream>
#include <vector>
#include <iomanip>
#include <iostream>
#include <sstream>

using namespace std;

__device__ void strcat_dev(char *dst, const char *src)
{
	int i = 0;
	for(; dst[i] != 0; i++);
	for(int j = 0; src[j] != 0; j++, i++)
		dst[i] = src[i];
	dst[++i] = 0;
}

struct Struct
{
    char pav[50];
    int kiekis;
    double kaina;
public:
	__host__ Struct(string pav, int kiekis, double kaina);
	__device__ Struct();
    string Print();
};

__host__ Struct::Struct(string pav, int kiekis, double kaina)
	:kiekis(kiekis), kaina(kaina)
{
	this->pav[0] = 0;
	strcat(this->pav, pav.c_str());
}

__device__ Struct::Struct()
	:kiekis(0), kaina(0.0)
{
	pav[0] = 0;
}

string Struct::Print()
{
        stringstream ss;
        ss << setw(15) << pav << setw(7) << kiekis << setw(20) << kaina;
        return ss.str();
}

__device__ Struct operator+(const Struct &left, const Struct &right)
{
	Struct ret;
	ret.kiekis = left.kiekis + right.kiekis;
	ret.kaina = left.kaina + right.kaina;
	strcat_dev(ret.pav, left.pav);
	strcat_dev(ret.pav, right.pav);
	return ret;
}

vector<vector<Struct>> ReadStuff(string file);
vector<string> ReadLines(string file);

string Titles();
string Print(int nr, Struct &s);
void syncOut(vector<vector<Struct>>&);


int main()
{
        auto input = ReadStuff("LapunasD.txt");
		int i = 0;

        cout << "\nsinchroninis isvedimas\n\n";
        syncOut(input);
        
		thrust::device_vector<int> devKeys;
		thrust::device_vector<Struct> devData;
        
        for(auto &vec : input)
        {
			for(int i = 0; i < vec.size(); i++)
            {
				devData.push_back(vec[i]);
				devKeys.push_back(i);
            }
        }

		thrust::sort_by_key(devKeys.begin(), devKeys.end(), devData.begin());

		thrust::device_vector<int> outputKeys;
		outputKeys.reserve(devKeys.size());
		thrust::device_vector<Struct> outputData;
		outputData.reserve(devKeys.size());

		thrust::equal_to<int> pred;
		thrust::plus<Struct> plus;

		try
		{
			thrust::reduce_by_key(devKeys.begin(), devKeys.end(), devData.begin(), outputKeys.begin(), outputData.begin(), pred, plus);
		}
		catch(thrust::system_error e)
		{
			cout << e.what() << endl;
			system("pause");
			return 1;
		}


		for(int i = 0; i < outputData.size(); i++)
		{
			Struct res = outputData[i];
			cout << setw(3) << i << setw(30) << res.pav << setw(7) << res.kiekis << setw(10) << res.kaina << endl;
		}

		system("pause");
		cout << i;

        return 0;
}

vector<vector<Struct>> ReadStuff(string file)
{
    auto lines = ReadLines(file);
    vector<vector<Struct>> ret;
    vector<Struct> tmp;
    for(size_t i = 0; i < lines.size(); i++)
    {
        if(lines[i] == "")
        {
            ret.push_back(move(tmp));
        }
        else
        {
			int start, end;
			start = 0;
			end = lines[i].find(' ');
			string pav = lines[i].substr(0, end);
			start = end + 1;
			end = lines[i].find(' ', start);
			int kiekis = stoi(lines[i].substr(start, end - start));
			start = end + 1;
			double kaina = stod(lines[i].substr(start));
			tmp.emplace_back(pav.c_str(), kiekis, kaina);
        }
    }
    return ret;
}

vector<string> ReadLines(string file)
{
        vector<string> ret;
        ifstream duom(file);
        while(!duom.eof())
        {
                string line;
                getline(duom, line);
                ret.push_back(line);
        }
        return ret;
}

string Titles()
{
        stringstream ss;
        ss << setw(15) << "Pavadiniams" << setw(7) << "Kiekis" << setw(20) << "Kaina";
        return ss.str();
}

void syncOut(vector<vector<Struct>> &data)
{
        cout << setw(3) << "Nr" << Titles() << endl << endl;
        for(size_t i = 0; i < data.size(); i++)
        {
                auto &vec = data[i];
                cout << "Masyvas" << i << endl;
                for(size_t j = 0; j < vec.size(); j++)
                {
                        cout << Print(j, vec[j]) << endl;
                }
        }
}

string Print(int nr, Struct &s)
{
        stringstream ss;
        ss << setw(3) << nr << s.Print();
        return ss.str();
}